#include <stdio.h>

using namespace std;

#include "StateSpace.cuh"

#include "Mesh.cuh"
#include "liley/Model.cuh"

int main(void)
{
	Mesh mesh(10, 10, 0.0001, 100);

	mesh.stepAndFlush(1, 0.0001, cout);
	mesh.stepAndFlush(1.0001, 0.0001, cout);

    hipDeviceSynchronize();
    printf("%s\n", hipGetErrorString( hipGetLastError() ) );
	return 0;
}
