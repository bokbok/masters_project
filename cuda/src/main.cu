#include "common.cuh"

using namespace std;

#include "liley/SIRU1Model.cuh"
#include "liley/SIRU2Model.cuh"
#include "liley/SIRU3Model.cuh"

#include "params/SIRU1HardcodedParams.cuh"
#include "params/SIRU2HardcodedParams.cuh"
#include "params/SIRU3HardcodedParams.cuh"
#include "SimulationRunner.cuh"

const char * OUTPUT_PATH = "/terra/runs";

int main(void)
{
	SIRU3HardcodedParams params;
	SimulationRunner<SIRU3Model> runner(params, OUTPUT_PATH);
	runner.runSimulation();

    hipDeviceSynchronize();
    printf("%s\n", hipGetErrorString( hipGetLastError() ) );
	return 0;
}
